#include "hip/hip_runtime.h"
/*
/*
 Random Faults in CUDA
 (C) Bedrich Benes 2020
 bbenes@purdue.edu
*/

#include "hip/hip_runtime.h"
#include ""

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>
#include <math.h>
#include <vector>			//Standard template library class
#include <GL/freeglut.h>

//in house created libraries
#include "vect3d.h"
#include "trackball.h"
#include "helper.h"         


#pragma comment(lib, "freeglut.lib")

TrackBallC trackball;
bool mouseLeft, mouseMid, mouseRight;

GLint wWindow = 1200;
GLint hWindow = 800;


#define DEBUG
const int MAX = 512;
const int SCENE = 1;
const int lineNum = 256;
const int LINE_MAX = 5000;

GLint n = 100;
GLfloat a[MAX][MAX];
GLint fill = 1;
#define ELEV 0.0005f

float lines[lineNum*3]; // a line is represented as y = k * x + b, k and b are stored
__constant__ float d_lines[LINE_MAX*3]; //allocates const memory

//CUDA stuff
float *d_A;

void Cleanup(bool noError)
{
	hipError_t error;
	// Free device memory
	if (d_A) error = hipFree(d_A);
	if (!noError || error != hipSuccess) printf("Something failed \n");
}

void InitLines() {

	for (int i = 0; i < lineNum; i++) {
		float k, b, dir;
		k = rand() / (float)(RAND_MAX) * 4 - 2;
		if (k > 0) b = rand() / (float)(RAND_MAX) * (k + 1) * MAX - k * MAX;
		else b = rand() / (float)(RAND_MAX) * (1 - k) * MAX;
		dir = rand() / (float)(RAND_MAX)-0.5;

		lines[i * 3] = k;
		lines[i * 3 + 1] = b;
		lines[i * 3 + 2] = dir > 0.0f ? 1 : 1;
	}
}

void RandomFaultsCuda();

void Idle(void)
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT); //clear all
	glEnable(GL_LIGHT0);
	trackball.Set3DViewCamera();
	glDisable(GL_LIGHTING);
	CoordSyst();
	glEnable(GL_LIGHTING);
	glCallList(SCENE);
	glutSwapBuffers();
}


Vect3d CrossProduct(Vect3d *a, Vect3d *b, Vect3d *cross)

{
	cross->SetX(a->GetY() * b->GetZ() - a->GetZ()* b->GetY());
	cross->SetY(a->GetZ() * b->GetX() - a->GetX()* b->GetZ());
	cross->SetZ(a->GetX() * b->GetY() - a->GetY()* b->GetX());
	return *cross;
}

void Display(void)
{
	int i, j;
	Vect3d v1, v2, v3, v13, v12, n;
	GLfloat materialColor[] = { 0.1f, 0.5f, 0.02f, 1.0f };
	GLfloat materialSpecular[] = { 0,0,0,1 };
	glNewList(SCENE, GL_COMPILE);
	glShadeModel(GL_SMOOTH);
	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE, materialColor);
	glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, materialSpecular);
	if (fill) glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	else glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	for (i = 0; i < MAX - 1; i++)
	{
		glBegin(GL_TRIANGLE_STRIP);
		for (j = 0; j < MAX-1; j++)
		{
//triangle 1
			v1.Set(i / (GLfloat)MAX, j / (GLfloat)MAX, a[i][j]);
			v2.Set((i + 1) / (GLfloat)MAX, j / (GLfloat)MAX, a[i + 1][j]);
			v3.Set((i + 1) / (GLfloat)MAX, (j + 1) / (GLfloat)MAX, a[i + 1][j+1]);
			v12.Set(v1.GetX() - v3.GetX(), v1.GetY() - v3.GetY(), v1.GetZ() - v3.GetZ());
			v13.Set(v1.GetX() - v2.GetX(), v1.GetY() - v2.GetY(), v1.GetZ() - v2.GetZ());
			n.Set(CrossProduct(&v13,&v12,&n));
			n.Normalize();
			glNormal3fv(n);
			glVertex3fv(v1);
			glVertex3fv(v2);
			glVertex3fv(v3);
//triangle 2
			v1.Set(i / (GLfloat)MAX, j / (GLfloat)MAX, a[i][j]);
			v2.Set((i + 1) / (GLfloat)MAX, (j + 1) / (GLfloat)MAX, a[i + 1][j+1]);
			v3.Set((i) / (GLfloat)MAX, (j + 1) / (GLfloat)MAX, a[i][j+1]);
			v12.Set(v1.GetX() - v3.GetX(), v1.GetY() - v3.GetY(), v1.GetZ() - v3.GetZ());
			v13.Set(v1.GetX() - v2.GetX(), v1.GetY() - v2.GetY(), v1.GetZ() - v2.GetZ());
			n.Set(CrossProduct(&v13, &v12, &n));
			n.Normalize();
			glNormal3fv(n);
			glVertex3fv(v1);
			glVertex3fv(v2);
			glVertex3fv(v3);
		}
		glEnd();
	}
	glEndList();
}

void DisplayUgly(void)
{
	int i, j;

	glNewList(SCENE, GL_COMPILE);
	glColor3ub(0, 0, 0);
	if (fill) glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	else glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	for (i = 0; i < MAX - 1; i++)
	{
		glBegin(GL_QUAD_STRIP);
		for (j = 0; j < MAX; j++)
		{
			glColor3f(a[i][j], a[i][j], a[i][j]);
			glVertex3f(i / (GLfloat)MAX, j / (GLfloat)MAX, a[i][j]);
			glVertex3f((i + 1) / (GLfloat)MAX, j / (GLfloat)MAX, a[i + 1][j]);
		}
		glEnd();
	}
	glEndList();
}

void Init(void)
{
	int i, j;

	glClearColor(1.0, 1.0, 1.0, 1.0);
	glClearDepth(1000.f);
	glEnable(GL_DEPTH_TEST);
	for (i = 0; i < MAX; i++)
		for (j = 0; j < MAX; j++)  a[i][j] = 0.5;

}


void myReshape(int w, int h)
{
	glViewport(0, 0, w, h);
	wWindow = w;
	hWindow = h;
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(-0.2, 1.2, -0.2, 1.2, -10, 10);
}


void RandomFault(int k)
{
	//Write the CPU version here
	for (int i = 0; i < MAX; i++) {
		for (int j = 0; j < MAX; j++) {
			if (j >= lines[k * 3] * i + lines[k * 3 + 1]) a[i][j] += lines[k * 3 + 2] * 0.001;
			else a[i][j] -= lines[k * 3 + 2] * 0.001;
		}
	}
}


void Key(unsigned char key, GLint i, GLint j)
{
	switch (key)
	{
	case 'f':
	case 'F': fill = (fill == 0); glutPostRedisplay(); break;
	case ' ': //run CPU implementation
	{
		InitLines();
		long t1 = clock();
		for (int i = 0; i < lineNum; i++)
		{
			char name[200];
			sprintf(name, "%i%% done\r", 100 * (i + 1) / lineNum);
			glutSetWindowTitle(name);
			RandomFault(i);
		}
		long t2 = clock();
		glutSetWindowTitle("Random Faults in Cuda");
		printf("CPU Running time: %i\n", t2 - t1);
		break;
	}
	case 'c': //run CUDA implementation
	{
		InitLines();
		glutSetWindowTitle("Running CUDA");

		hipEvent_t startT, stopT;
		float time;

		hipEventCreate(&startT);
		hipEventCreate(&stopT);
		hipEventRecord(startT, 0);

		RandomFaultsCuda();

		hipEventRecord(stopT, 0);
		hipEventSynchronize(stopT);
		hipEventElapsedTime(&time, startT, stopT);
		hipEventDestroy(startT);
		hipEventDestroy(stopT);

		glutSetWindowTitle("Random Faults in Cuda");
		printf("CUDA Running time: %f\n", time);
		break;
	}
	case 27:
	case 'q':
	case 'Q': exit(0);
	}
	glutPostRedisplay();
}

void Mouse(int button, int state, int x, int y) {
	if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
	{
		trackball.Set(true, x, y);
		mouseLeft = true;
	}
	if (button == GLUT_LEFT_BUTTON && state == GLUT_UP)
	{
		trackball.Set(false, x, y);
		mouseLeft = false;
	}
	if (button == GLUT_MIDDLE_BUTTON && state == GLUT_DOWN)
	{
		trackball.Set(true, x, y);
		mouseMid = true;
	}
	if (button == GLUT_MIDDLE_BUTTON && state == GLUT_UP)
	{
		trackball.Set(true, x, y);
		mouseMid = false;
	}
	if (button == GLUT_RIGHT_BUTTON && state == GLUT_DOWN)
	{
		trackball.Set(true, x, y);
		mouseRight = true;
	}
	if (button == GLUT_RIGHT_BUTTON && state == GLUT_UP)
	{
		trackball.Set(true, x, y);
		mouseRight = false;
	}
}

void MouseMotion(int x, int y) {
	if (mouseLeft)  trackball.Rotate(x, y);
	if (mouseMid)   trackball.Translate(x, y);
	if (mouseRight) trackball.Zoom(x, y);
//	glutPostRedisplay();
}

__global__ void RandFaultKernel(float a[MAX][MAX],  //2D array of elements
	const int N, //array is N*N
	const int lineNum) //number of steps to run
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;

	for (int k = 0; k < lineNum; k++) {
		if (j > d_lines[k * 3] * i + d_lines[k * 3 + 1]) a[i][j] += d_lines[k*3+2] * 0.001;
		else a[i][j] -= d_lines[k * 3 + 2] * 0.001;
	}
}


void RandomFaultsCuda()
{

	//printf("Constant memory copyed\n");
	hipError_t error;
	int sizeArray;

	//allocate array on the device
	sizeArray = sizeof(float)*MAX*MAX; //2D array of floats
	error = hipMalloc((void**)&d_A, sizeArray);
	//Copy the 2D array from host memory to device memory
	error = hipMemcpy(d_A, a, sizeArray, hipMemcpyHostToDevice);
	if (error != hipSuccess) Cleanup(false);

	//Copy line messages to constant memory
	error = hipMemcpyToSymbol(HIP_SYMBOL(d_lines), lines, lineNum * 3 * sizeof(float)); //copy values
	if (error != hipSuccess) printf("Constant memory copy fault: %i\n", error);

	//printf("Constant memory copyed\n");

	//prepare blocks and grid
	const int BLOCKSIZE = 16;
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid(ceil((float)MAX / dimBlock.x),
		         ceil((float)MAX / dimBlock.y));
	// Invoke kernel
	RandFaultKernel << <dimGrid, dimBlock >> > ((float(*)[MAX])d_A, MAX, lineNum);
	error = hipGetLastError();
	if (error != hipSuccess) printf("Something went wrong: %i\n", error);
	error = hipDeviceSynchronize();
	if (error != hipSuccess) { printf("synchronization is wrong\n"); Cleanup(false); }
	// Copy result from device memory to host memory
	error = hipMemcpy(a, d_A, sizeArray, hipMemcpyDeviceToHost);
	if (error != hipSuccess) { printf("could not copy from device\n"); Cleanup(false); }
	Cleanup(true);
}

// Host code
int main(int argc, char** argv)
{
	srand(5);
	glutInitWindowSize(wWindow, hWindow);
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	glutCreateWindow("Random Faults");
	Init();
	glutDisplayFunc(Display);
	glutIdleFunc(Idle);
	glutKeyboardFunc(Key);
	glutReshapeFunc(myReshape);
	glutMouseFunc(Mouse);
	glutMotionFunc(MouseMotion);
	glutMainLoop();
	return 0;

}


